#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "CudaFunctions.cuh"


__global__ void GetVoxelsAnatomicalSegmentationKernel(unsigned char* anatomicalImage, unsigned char* segmentedImage, SegmentData* segmentationTable, int segmentsCount, unsigned char* segmentationTransferTable, int eps, RGBVoxel* voxels, int width, int height, int curDepth, int curNumber, int* count)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	if (x >= width || y >= height)
	{
		return;
	}
	int curPos = (y*width + x) * 3;
	int4 pixel;
	pixel.z = segmentedImage[curPos];
	pixel.y = segmentedImage[curPos + 1];
	pixel.x = segmentedImage[curPos + 2];
	if ((pixel.x == segmentationTable[0].color.x) && (pixel.y == segmentationTable[0].color.y) && (pixel.z == segmentationTable[0].color.z))
	{
		return;
	}
	for (int i = 1; i < segmentsCount; i++)
	{
		if ((curDepth >= segmentationTable[i].start) && (curDepth <= segmentationTable[i].finish))
		{
			if (abs(pixel.x - segmentationTable[i].color.x)<eps && abs(pixel.y - segmentationTable[i].color.y)<eps && abs(pixel.z - segmentationTable[i].color.z)<eps)
			{
				if (segmentationTransferTable[i] == 0)
				{
					return;
				}
				int curCount = atomicAdd(count, 1);
				voxels[curCount].color.w = segmentationTransferTable[i];
				voxels[curCount].color.z = anatomicalImage[curPos];
				voxels[curCount].color.y = anatomicalImage[curPos + 1];
				voxels[curCount].color.x = anatomicalImage[curPos + 2];
				voxels[curCount].index = width * height * curNumber + width * y + x;
				return;
			}
		}
	}
}

void GetVoxelsAnatomicalSegmentation(unsigned char* anatomicalImage, unsigned char* segmentedImage, SegmentData* segmentationTable, int segmentsCount, unsigned char* segmentationTransferTable, int eps, RGBVoxel* voxels, int width, int height, int curDepth, int curNumber, int* count)
{
	dim3 blockSize(32, 32);
	dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);
	GetVoxelsAnatomicalSegmentationKernel <<<gridSize, blockSize >>> (anatomicalImage, segmentedImage, segmentationTable, segmentsCount, segmentationTransferTable, eps, voxels, width, height, curDepth, curNumber, count);
}

struct CompareVoxelsRed
{
	__host__ __device__ bool operator()(const RGBVoxel &a, const RGBVoxel &b) { return (a.color.x < b.color.x); };
};
struct CompareVoxelsGreen
{
	__host__ __device__ bool operator()(const RGBVoxel &a, const RGBVoxel &b) { return (a.color.y < b.color.y); };
};
struct CompareVoxelsBlue
{
	__host__ __device__ bool operator()(const RGBVoxel &a, const RGBVoxel &b) { return (a.color.z < b.color.z); };
};
struct CompareVoxelsIndex
{
	__host__ __device__ bool operator()(const RGBVoxel &a, const RGBVoxel &b) { return (a.index < b.index); };
};

struct RGBToPalette
{
	int paletteIndex;
	RGBToPalette(int _palettedIndex)
	{
		paletteIndex = _palettedIndex;
	}
	__host__ __device__ RGBVoxel operator()(const RGBVoxel &a)
	{ 
		RGBVoxel result;
		result.index = a.index;
		result.color.x = paletteIndex;
		result.color.y = a.color.w;
		result.color.z = 0;
		result.color.w = 0;
		return result;
	};
};

struct ReduceColors
{
	__host__ __device__ ulonglong4 operator()(const RGBVoxel &a, const RGBVoxel &b)
	{ 
		ulonglong4 res;
		res.x = a.color.x + b.color.x;
		res.y = a.color.y + b.color.y;
		res.z = a.color.z + b.color.z;
		return res; 
	};
};

void CUDACreateFromSlices(string anatomicalFolder, string segmentedFolder, vector<SegmentData>& segmentationTable, vector<unsigned char>& segmentationTransfer, int eps, int3& dim, vector<Voxel>& voxels, vector<uchar4>& palette)
{

	HANDLE hA;
	WIN32_FIND_DATAA fA;
	HANDLE hS;
	WIN32_FIND_DATAA fS;
	vector<string> filesA;
	vector<string> filesS;
	hA = FindFirstFileA((anatomicalFolder + "*").c_str(), &fA);  //Find "."
	FindNextFileA(hA, &fA); //Find ".."
	FindNextFileA(hA, &fA); //Find real filename
	Mat img = imread(anatomicalFolder + fA.cFileName);
	dim.x = img.size().width;
	dim.y = img.size().height;
	dim.z = 1;
	while (FindNextFileA(hA, &fA))
	{
		dim.z++;
	}
	hA = FindFirstFileA((anatomicalFolder + "*").c_str(), &fA);  //Find "."
	FindNextFileA(hA, &fA); //Find ".."
	hS = FindFirstFileA((segmentedFolder + "*").c_str(), &fS);  //Find "."
	FindNextFileA(hS, &fS); //Find ".."
	int curDepth = 0;
	int curNumber = 0;
	unsigned char* hDataA;
	unsigned char* hDataS;
	unsigned char* gDataA;
	unsigned char* gDataS;
	SegmentData* gSegmentTable;
	unsigned char* gSegmentationTransfer;
	thrust::device_vector<RGBVoxel> dVoxelsSlice(dim.x*dim.y);
	thrust::device_vector<RGBVoxel> dVoxels;
	int* gCount;
	int hCount;
	hipMalloc((void**)&gDataA, sizeof(unsigned char)*(dim.x*dim.y) * 3);
	hipMalloc((void**)&gDataS, sizeof(unsigned char)*(dim.x*dim.y) * 3);
	hipMalloc((void**)&gSegmentTable, sizeof(SegmentData)*(segmentationTable.size()));
	hipMemcpy(gSegmentTable, &segmentationTable[0], sizeof(SegmentData)*(segmentationTable.size()), hipMemcpyHostToDevice);
	hipMalloc((void**)&gSegmentationTransfer, sizeof(SegmentData)*(segmentationTransfer.size()));
	hipMemcpy(gSegmentationTransfer, &segmentationTransfer[0], sizeof(SegmentData)*(segmentationTransfer.size()), hipMemcpyHostToDevice);
	hipMalloc((void**)&gCount, sizeof(int));
	Mat mA;
	Mat mS;
	while (FindNextFileA(hA, &fA) && FindNextFileA(hS, &fS))
	{
		mA = imread(anatomicalFolder + fA.cFileName);
		mS = imread(segmentedFolder + fS.cFileName);
		curDepth = atoi(fA.cFileName);
		hipMemset(gCount, 0, sizeof(int));
		hDataA = mA.data;
		hDataS = mS.data;
		hipMemcpy(gDataA, hDataA, sizeof(unsigned char)*(dim.x*dim.y) * 3, hipMemcpyHostToDevice);
		hipMemcpy(gDataS, hDataS, sizeof(unsigned char)*(dim.x*dim.y) * 3, hipMemcpyHostToDevice);
		GetVoxelsAnatomicalSegmentation(gDataA, gDataS, gSegmentTable, segmentationTable.size(), gSegmentationTransfer, eps, thrust::raw_pointer_cast(dVoxelsSlice.data()), dim.x, dim.y, curDepth, curNumber, gCount);
		hipMemcpy(&hCount, gCount, sizeof(int), hipMemcpyDeviceToHost);
		if (hCount > 0)
		{
			int curSize = dVoxels.size();
			dVoxels.resize(curSize + hCount);
			thrust::copy(dVoxelsSlice.begin(), dVoxelsSlice.begin()+hCount, dVoxels.begin()+ curSize);
		}
		curNumber++;
	}
	hipFree(gDataA);
	hipFree(gDataS);
	hipFree(gSegmentTable);
	hipFree(gSegmentationTransfer);
	hipFree(gCount);

	queue<PaletteElement> qPalette;
	qPalette.emplace(dVoxels.size());
	vector<PaletteElement> finalPaletteElements;
	RGBVoxel min;
	RGBVoxel max;
	int len = 0;

	while (!qPalette.empty())
	{
		PaletteElement cur = qPalette.front();
		min = *thrust::min_element(dVoxels.begin() + cur.start, dVoxels.begin() + cur.start + cur.length, CompareVoxelsRed());
		max = *thrust::max_element(dVoxels.begin() + cur.start, dVoxels.begin() + cur.start + cur.length, CompareVoxelsRed());
		len = max.color.x - min.color.x;
		cur.sortMode = PaletteElement::SORT_MODE::SORT_MODE_RED;
		min = *thrust::min_element(dVoxels.begin() + cur.start, dVoxels.begin() + cur.start + cur.length, CompareVoxelsGreen());
		max = *thrust::max_element(dVoxels.begin() + cur.start, dVoxels.begin() + cur.start + cur.length, CompareVoxelsGreen());
		if (len < (max.color.y - min.color.y))
		{
			len = max.color.y - min.color.y;
			cur.sortMode = PaletteElement::SORT_MODE::SORT_MODE_GREEN;
		}	
		min = *thrust::min_element(dVoxels.begin() + cur.start, dVoxels.begin() + cur.start + cur.length, CompareVoxelsBlue());
		max = *thrust::max_element(dVoxels.begin() + cur.start, dVoxels.begin() + cur.start + cur.length, CompareVoxelsBlue());
		if (len < (max.color.z - min.color.z))
		{
			len = max.color.z - min.color.z;
			cur.sortMode = PaletteElement::SORT_MODE::SORT_MODE_BLUE;
		}
		switch (cur.sortMode)
		{
		case PaletteElement::SORT_MODE::SORT_MODE_RED:
			thrust::sort(dVoxels.begin() + cur.start, dVoxels.begin() + cur.start + cur.length, CompareVoxelsRed());
			break;
		case PaletteElement::SORT_MODE::SORT_MODE_GREEN:
			thrust::sort(dVoxels.begin() + cur.start, dVoxels.begin() + cur.start + cur.length, CompareVoxelsGreen());
			break;
		case PaletteElement::SORT_MODE::SORT_MODE_BLUE:
			thrust::sort(dVoxels.begin() + cur.start, dVoxels.begin() + cur.start + cur.length, CompareVoxelsBlue());
			break;
		}
		qPalette.pop();
		if (cur.level == 8)
		{
			finalPaletteElements.push_back(cur);
		}
		else
		{
			qPalette.emplace(cur, true);
			qPalette.emplace(cur, false);
		}
	}
	thrust::host_vector<RGBVoxel> hVoxels = dVoxels;
	vector<pair<uchar4, int>> tmpPalette;
	for (int i = 0; i < finalPaletteElements.size(); i++)
	{
		UINT64 r = 0;
		UINT64 g = 0;
		UINT64 b = 0;
		for (int j = finalPaletteElements[i].start; j < finalPaletteElements[i].start + finalPaletteElements[i].length; j++)
		{
			r += hVoxels[j].color.x;
			g += hVoxels[j].color.y;
			b += hVoxels[j].color.z;
		}
		uchar4 color = { r / finalPaletteElements[i].length,g / finalPaletteElements[i].length,b / finalPaletteElements[i].length,0 };
		tmpPalette.emplace_back(color, i);		
	}
	hVoxels.clear();
	hVoxels.shrink_to_fit();
	std::sort(tmpPalette.begin(), tmpPalette.end(), [](auto &a, auto&b) {return CompareColorsIntensity(a.first, b.first); });
	for (int i = 0; i < tmpPalette.size(); i++)
	{
		palette.push_back(tmpPalette[i].first);
		RGBToPalette curTransform(i);
		thrust::transform(dVoxels.begin() + finalPaletteElements[tmpPalette[i].second].start, dVoxels.begin() + finalPaletteElements[tmpPalette[i].second].start + finalPaletteElements[tmpPalette[i].second].length, dVoxels.begin() + finalPaletteElements[tmpPalette[i].second].start, curTransform);
	}
	thrust::sort(dVoxels.begin(), dVoxels.end(), CompareVoxelsIndex());
	voxels.resize(dVoxels.size());
	hipMemcpy(&voxels[0], thrust::raw_pointer_cast(dVoxels.data()), sizeof(Voxel)*voxels.size(), hipMemcpyDeviceToHost);
}

__device__ void SetMaskElement(uint index, int* mask)
{
	int pos = 1 << (index % 32);
	atomicOr(mask[index / 32], pos);
}

__global__ void CalculateIntersectingVoxelsKernel(Voxel* voxels, uint3 voxelsDim, int voxelsCount, ushort2* dist, Vector3 invDir, Vector3 dirOrigin, uint boneIndex, int* mask)
{
	uint x = blockIdx.x*blockDim.x + threadIdx.x;
	uint y = blockIdx.y*blockDim.y + threadIdx.y;
	uint index = y * gridDim.x * blockDim.x + x;
	if (index >= voxelsCount)
	{
		return;
	}
	if ((dist[index].x == 0) && (dist[index].y == 0))
	{
		return;
	}
	uint voxIndex = voxels[index].index;
	Vector3 voxPos;
	voxPos.z = voxIndex / (voxelsDim.y * voxelsDim.x);
	uint tmp = voxIndex % (voxelsDim.y * voxelsDim.x);
	voxPos.y = tmp / voxelsDim.x;
	voxPos.x = tmp % voxelsDim.x;
	///Intersecting
	float tmin = (voxPos.x - dirOrigin.x) * invDir.x;
	float tmax = (voxPos.x + 1 - dirOrigin.x) * invDir.x;
	float t1min = (voxPos.y - dirOrigin.y) * invDir.y;
	float t1max = (voxPos.y + 1 - dirOrigin.y) * invDir.y;
	if ((tmin > t1max) || (t1min > tmax))
	{
		return;
	}
	if (t1min > tmin)
	{
		tmin = t1min;
	}
	if (t1max < tmax)
	{
		tmax = t1max;
	}
	t1min = (voxPos.z - dirOrigin.z) * invDir.z;
	t1max = (voxPos.z + 1 - dirOrigin.z) * invDir.z;
	if ((tmin > t1max) || (t1min > tmax))
	{
		return;
	}
	if (t1min > tmin)
	{
		tmin = t1min;
	}
	if (t1max < tmax)
	{
		tmax = t1max;
	}
	if (((tmin >= 0) && (tmin <= 1)) || ((tmax >= 0) && (tmax <= 1)))
	{
		if (atomicMin(dist[index].x, 0) != 0)
		{
			voxels[index].bone01 = boneIndex;
			SetMaskElement(index, mask);
		}
		else if (atomicMin(dist[index].y, 0) != 0)
		{
			voxels[index].bone02 = boneIndex;
			SetMaskElement(index, mask);
		}
	}
}

void CalculateIntersectingVoxelsKernel(Voxel* voxels, uint3 voxelsDim, int voxelsCount, ushort2* dist, Vector3 invDir, Vector3 dirOrigin, uint boneIndex, int* mask)
{
	dim3 blockSize(32, 32);
	int computeBlocksCount = ceil(sqrt(voxelsCount));
	computeBlocksCount = ceil(computeBlocksCount / 32.0);
	dim3 gridSize(computeBlocksCount, computeBlocksCount);
	CalculateIntersectingVoxelsKernel << <gridSize, blockSize >> > CalculateIntersectingVoxelsKernel(voxels, voxelsDim, voxelsCount, dist, invDir, dirOrigin, boneIndex, mask);
}

void CUDACalculateWeights(vector<Voxel>& voxels, uint3 voxelsDim, vector<float>& weights, vector<pair<Vector3,Vector3>>& bonesPoints)
{
	int* dMask00;
	int* dMask01;
	int bitsetSize = (voxels.size() - 1) / (sizeof(int) * 8) + 1;
	hipMalloc((void**)&dMask00, sizeof(int) * bitsetSize);
	hipMalloc((void**)&dMask01, sizeof(int) * bitsetSize);
	hipMemset((void**)&dMask00, 0, sizeof(int) * bitsetSize);
	hipMemset((void**)&dMask01, 0, sizeof(int) * bitsetSize);
	thrust::device_vector<ushort2> dDist(voxels.size(), { USHRT_MAX, USHRT_MAX });
	thrust::device_vector<Voxel> dVoxels(voxels.begin(), voxels.end());
	for (int i = 0; i < bonesPoints.size(); i++)
	{
		Vector3 invDir = bonesPoints[i].second - bonesPoints[i].first;
		invDir.x = 1.0f / invDir.x;
		invDir.y = 1.0f / invDir.y;
		invDir.z = 1.0f / invDir.z;
		CalculateIntersectingVoxelsKernel(thrust::raw_pointer_cast(dVoxels), voxelsDim, voxels.size(), thrust::raw_pointer_cast(dDist), invDir, bonesPoints[i].first, i, dMask00);
	}
}
